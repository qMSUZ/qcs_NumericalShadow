#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#include "cplxNum.h"
#include "samplingroutine.h"


#ifndef min
#define min(x,y) (x>y?x:y)
#endif


void sampling_routine_test_simpleComplex_float()
{
	const int vector_dim = 4;
	const int sampling_points = 8;

	simpleComplexFloat *host_mat;
	simpleComplexFloat *host_pts;

	int i;

	host_mat = (simpleComplexFloat*)malloc( sizeof( simpleComplexFloat ) * vector_dim * vector_dim );
	host_pts = (simpleComplexFloat*)malloc( sizeof( simpleComplexFloat ) * sampling_points );

	/* definition of test matrix
	m = [ -0.736007-0.486131im   -0.50265+0.303371im   -1.13945-0.492444im    0.521185+1.68789im ;
		   0.058992-0.682839im   -0.24367+0.851119im    1.63444+1.08207im     0.471581-0.283695im ;
           0.427102+1.99722im     0.0900991-2.27228im   0.294868+0.495371im  -1.07317-0.438353im ;
           0.782054-1.08824im     0.261584-0.122245im   0.921061+0.306364im   0.456282+2.01316im ]
	*/

	host_mat[0]  = make_simpleComplexFloat(-0.736007f, -0.486131f);
	host_mat[1]  = make_simpleComplexFloat(-0.50265f,   0.303371f);
	host_mat[2]  = make_simpleComplexFloat(-1.13945f,  -0.492444f);
	host_mat[3]  = make_simpleComplexFloat( 0.521185f,  1.68789f);

	host_mat[4]  = make_simpleComplexFloat( 0.058992f, -0.682839f);
	host_mat[5]  = make_simpleComplexFloat(-0.24367f,   0.851119f);
	host_mat[6]  = make_simpleComplexFloat( 1.63444f,   1.08207f);
	host_mat[7]  = make_simpleComplexFloat( 0.471581f, -0.283695f);

	host_mat[8]  = make_simpleComplexFloat( 0.427102f,  1.99722f);
	host_mat[9]  = make_simpleComplexFloat( 0.0900991f,-2.27228f);
	host_mat[10] = make_simpleComplexFloat( 0.294868f,  0.495371f);
	host_mat[11] = make_simpleComplexFloat(-1.07317f,  -0.438353f);

	host_mat[12] = make_simpleComplexFloat( 0.782054f, -1.08824f);
	host_mat[13] = make_simpleComplexFloat( 0.261584f, -0.122245f);
	host_mat[14] = make_simpleComplexFloat( 0.921061f,  0.306364f);
	host_mat[15] = make_simpleComplexFloat( 0.456282f,  2.01316f);

	//printf("\n");
	//print_square_matrix(host_mat, vector_dim);
	col_and_rows_swap_square_matrix(host_mat, vector_dim);
	//printf("transpose\n");
	//print_square_matrix(host_mat, vector_dim);

	/*
	column and rows swapped
	m = [ -0.736007-0.486131im  0.058992-0.682839im   0.427102+1.99722im   0.782054-1.08824im ;
		-0.50265+0.303371im  -0.24367+0.851119im  0.0900991-2.27228im   0.261584-0.122245im ;
		-1.13945-0.492444im   1.63444+1.08207im    0.294868+0.495371im  0.921061+0.306364im ;
		0.521185+1.68789im   0.471581-0.283695im   -1.07317-0.438353im  0.456282+2.01316im ]
	*/

	/*
	host_mat[0]  = make_simpleComplexFloat(-0.736007f, -0.486131f);
	host_mat[1]  = make_simpleComplexFloat( 0.058992f, -0.682839f);
	host_mat[2]  = make_simpleComplexFloat( 0.427102f,  1.99722f);
	host_mat[3]  = make_simpleComplexFloat( 0.782054f, -1.08824f);
	host_mat[4]  = make_simpleComplexFloat(-0.50265f,   0.303371f);
	host_mat[5]  = make_simpleComplexFloat(-0.24367f,   0.851119f);
	host_mat[6]  = make_simpleComplexFloat( 0.0900991f,-2.27228f);
	host_mat[7]  = make_simpleComplexFloat( 0.261584f, -0.122245f);
	host_mat[8]  = make_simpleComplexFloat(-1.13945f,  -0.492444f);
	host_mat[9]  = make_simpleComplexFloat( 1.63444f,   1.08207f);
	host_mat[10] = make_simpleComplexFloat( 0.294868f,  0.495371f);
	host_mat[11] = make_simpleComplexFloat( 0.921061f,  0.306364f);
	host_mat[12] = make_simpleComplexFloat( 0.521185f,  1.68789f);
	host_mat[13] = make_simpleComplexFloat( 0.471581f, -0.283695f);
	host_mat[14] = make_simpleComplexFloat(-1.07317f,  -0.438353f);
	host_mat[15] = make_simpleComplexFloat( 0.456282f,  2.01316f);
	*/

	//printf("vectors_set:\n");
	// int k;
	for (i = 0; i < sampling_points; i++)
	{
		/*
		for (k = 0; k < vector_dim; k++)
		{
			host_vectors_set[k + (i*vector_dim)] = 0.0f;
			host_tmpvs[k + (i*vector_dim)] = 0.0f;
		}
		*/
		host_pts[i].x = 1.0f;
		host_pts[i].y = 0.0f;
	}

 	int error_level = sampling_routine_simpleComplex_float(host_mat, host_pts, vector_dim, sampling_points, 0);

	printf("pts=[\n");
	for (i = 0; i < sampling_points; i++)
	{
		printf("%f+%fim ;\n", host_pts[i].x, host_pts[i].y);
	}
	printf("]\n");

	free((void*)host_pts);
	free((void*)host_mat);
}

int main(int argc, char *argv[])
{
	init_enviroment();

	sampling_routine_test_simpleComplex_float();

	/*
	simpleComplexFloat a, b, c;

	a = make_simpleComplexFloat(1.0f, 0.5f);
	b = make_simpleComplexFloat(0.25f, 0.75f);
	c = make_simpleComplexFloat(0.0f, 0.0f);

	c = a + b;
	*/

	finalize_environment();

	return 0;
}
